#ifndef GEMM_KERNEL_CUH
#define GEMM_KERNEL_CUH


// https://developer.nvidia.com/blog/cutlass-linear-algebra-cuda/
// https://github.com/NVIDIA/cutlass/blob/main/media/docs/fundamental_types.md


#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include "cutlass/gemm/device/gemm.h"


int accumulate_hessian_kernel(
    void* mat_hessian,
    const void* mat_input,
    const int size_hidden,  // number of rows of A and C matrices
    const int size_batch  // number of columns of A and rows of B matrices
) {
    // Define the GEMM type
    using Gemm = cutlass::gemm::device::Gemm<
        cutlass::half_t,
        cutlass::layout::ColumnMajor,
        cutlass::half_t,
        cutlass::layout::RowMajor,
        float,
        cutlass::layout::RowMajor
    >;

    // Create GEMM operation
    Gemm gemm_op;

    // Define GEMM arguments
    typename Gemm::Arguments args(
        {size_hidden, size_hidden, size_batch},  // Problem size
        {(cutlass::half_t*) mat_input, size_hidden},  // Tensor A (with leading dimension)
        {(cutlass::half_t*) mat_input, size_hidden},  // Tensor B (with leading dimension)
        {(float*) mat_hessian, size_hidden},  // Tensor C (output with leading dimension)
        {(float*) mat_hessian, size_hidden},  // Tensor D (same as C)
        {1.f, 1.f}  // scalars for alpha and beta
    );

    // Run the GEMM operation
    cutlass::Status status = gemm_op(args);

    if (status != cutlass::Status::kSuccess) {
        std::cerr << "GEMM operation failed: " << cutlass::cutlassGetStatusString(status) << std::endl;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}


#endif
